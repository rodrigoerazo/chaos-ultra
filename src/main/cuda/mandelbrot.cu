#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "math.h"
#include "helpers.hpp"
#include "float.h"

typedef unsigned int uint;
using Pointf = Point<float>;
using Pointi = Point<uint>;

__device__ const float PI_F = 3.14159265358979f;
__device__ const uint MAX_SS_LEVEL = 256;

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
  printf("Mandelbrot: Error at %s:%d\n",__FILE__,__LINE__); \
  return EXIT_FAILURE;}} while(0)
  
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
  printf("Mandelbrot: Error at %s:%d\n",__FILE__,__LINE__); \
  return EXIT_FAILURE;}} while(0)

#define DEBUG_MODE
#ifdef DEBUG_MODE 
  #define ASSERT(x) assert(x)
#else 
  #define ASSERT(x) do {} while(0)
#endif

#ifndef CUDART_VERSION
  #error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 9000) //for cuda 9 and later, use __any_sync(__activemask(), predicate) instead, see Programming guide, B.13 for more details
  #define __ALL(predicate) __all_sync(__activemask(), predicate)
  #define __ANY(predicate) __any_sync(__activemask(), predicate)
#else
  #define __ALL(predicate) __all(predicate)
  #define __ANY(predicate) __any(predicate)
#endif



//Mandelbrot content, using standard mathematical terminology for Mandelbrot set definition, i.e.
//  f_n = f_{n-1}^2 + c
//  f_0 = 0
//  thus iteratively applying: f(z) = z*z * c
//  where z, c are complex numbers, with components denoted as
//    x ... for real part (corresponding to geometric x-axis)
//    y ... for imag part (corresponding to geometric y-axis)

template <class Real> __device__ __forceinline__
uint escape(uint maxIterations, Point<Real> c){
  Point<Real> z(0,0);
  Real zx_new;
  uint i = 0;
  while(i < maxIterations && z.x*z.x+z.y*z.y < 4){
      zx_new = z.x*z.x-z.y*z.y + c.x;
      z.y = 2*z.x*z.y + c.y; 
      z.x = zx_new;
      ++i;
  }
  return i;
}


/// Dispersion in this context is "Index of dispersion", aka variance-to-mean ratio. See https://en.wikipedia.org/wiki/Index_of_dispersion for more details
template <class Real> __device__ __forceinline__
Real computeDispersion(uint* data, uint dataLength, Real mean){
  uint n = dataLength;
  Real variance = 0;
  for(uint i = 0; i < dataLength; i++){
    //using numerically stable Two-Pass algorithm, https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Two-pass_algorithm
    variance += (data[i]-mean)*(data[i]-mean);
  }
  variance /= (n-1); 
  return variance / mean;
}


__device__ __forceinline__
bool isWithinRadius(uint idx_x, uint idx_y, uint width, uint height, uint radius, uint focus_x, uint focus_y){
  if(__sad(idx_x, focus_x, 0) > radius / 2) return false;
  if(__sad(idx_y, focus_y, 0) > radius / 2) return false;
  else return true;

  // if(idx_x < (width - radius)/2 || idx_y < (height-radius)/2) return false;
  // if(idx_x > (width + radius)/2 || idx_y > (height+radius)/2) return false;
  // else return true;
}

__device__ long seed;
/// Intended for debugging only
__device__ __forceinline__ uint simpleRandom(uint val){
    long long a = 1103515245;
    long long c = 12345;
    long long m = 4294967295l; //2**32 - 1
    seed = (a * (val+seed) + c) % m;
    return seed;
}

__device__ const uint WARP_SIZE_X = 8; //represents desired size of the recatangular warp (with respect to threadIdx). WARP_SIZE.x * WARP_SIZE.y should always be warpSize (32 for CUDA 9 and lower)
__device__ const uint WARP_SIZE_Y = 4;
  /// Computes indexes to a per-pixel acces of a 2D image, based on threadIdx and blockIdx.
  /// Morover, threads in a warp will be arranged in a rectangle (rather than in single line as with the naive implementation).
  /// The caller should always check if the returned value exceeded image width and height.
__device__ const Point<uint> getImageIndexes(){
  const uint threadID = threadIdx.x + threadIdx.y * blockDim.x;
  const uint warpH = WARP_SIZE_Y; // 2,4,8 are only reasonable values of warpH for the following formula
  const uint blockWidth = blockDim.x * warpH;
  ASSERT (blockDim.x == 32); //following formula works only when blockDim.x is 32 
  const uint inblock_idx_x = (-(threadID % (warpH * warpH)) + threadID % blockWidth) / warpH + threadID % warpH;
  const uint inblock_idx_y = (threadID / blockWidth) * warpH + (threadID / warpH) % warpH;
  const uint idx_x = blockDim.x * blockIdx.x + inblock_idx_x;
  const uint idx_y = blockDim.y * blockIdx.y + inblock_idx_y;
  // { //debug
  //   uint warpid = threadID / warpH;
  //   if(idx_x < 20 && idx_y < 20){
  //     //printf("bw:%u\n", blockWidth);
  //     printf("%u\t%u\t%u\t%u\t%u\n", threadIdx.x, threadIdx.y, threadID ,inblock_idx_x, inblock_idx_y);
  //   }
  // }
  return Point<uint>(idx_x, idx_y);
}

__device__ __forceinline__
pixel_info_t* getPtrToPixel(pixel_info_t** array2D, long pitch, uint x, uint y){
  return (((pixel_info_t*)((char*)array2D + y * pitch)) + x);
}

/// param sampleCount: Maximum number of samples to take. Actual number of samples taken will be stored here before returning. If adaptiveSS==false, the value will not change.
template <class Real> __device__
uint sampleTheFractal(Pointi pixel, Pointi size, Rectangle<Real> image, uint maxIterations,uint & sampleCount, bool adaptiveSS){
  const uint adaptiveTreshold = 10;
  uint r[adaptiveTreshold];

  //We are in a complex plane from (left_bottom) to (right_top), so we scale the pixels to it
  Point<Real> pixelSize = image.size() / size.cast<Real>();
  
  uint escapeTimeSum = 0;
  ASSERT (sampleCount <= MAX_SS_LEVEL);
  for(uint i = 0; i < sampleCount; i++){
    Point<Real> delta = Point<Real>(i / (Real) sampleCount);
    
    // c = {LBx, RTy} {+,-} ((pixel+delta) * pixelSize)
    const Point<Real> c = Point<Real>(image.left_bottom.x, image.right_top.y) +
      Point<Real>(1,-1) * (pixel.cast<Real>() + delta) * pixelSize;

    uint escapeTime = escape(maxIterations, c);
    escapeTimeSum += escapeTime;
    if(i < adaptiveTreshold){
      r[i] = escapeTime;
    }

    //todo this process should ideally adaptive, at least dynamic. Not predefined like.
    if(( i== 2 || i == adaptiveTreshold) && adaptiveSS){ //decide whether to continue with supersampling or not
      Real mean = escapeTimeSum / (i+1);
      Real dispersion = computeDispersion(r, i, mean);
      float dispersionMax = (i==2) ? 0.01 : 0.1;
      if(__ALL(dispersion <= dispersionMax )){
        sampleCount = i+1; //effectively disabling high SS and storing info about actual number of samples taken
      }
      //else we are on an chaotic edge, thus as many samples as possible are needed
    }
  }
  uint result = escapeTimeSum / sampleCount; 
  return result;
   

  //debug:
  // if(idx_x < 10 && idx_y < 10){
  //   printf("%f\t", randomSample);
  //   __syncthreads();
  //   if(idx_x == 0 && idx_y == 0)
  //     printf("\n");
  // }
} 

__device__ const uint USE_ADAPTIVE_SS_FLAG_MASK = (1 << 0);
__device__ const uint VISUALISE_SAMPLE_COUNT_FLAG_MASK = (1 << 1);
__device__ const uint USE_FOVEATION_FLAG_MASK = (1 << 2);
__device__ const uint USE_SAMPLE_REUSE_FLAG_MASK = (1 << 3);
__device__ const uint IS_ZOOMING_FLAG_MASK = (1 << 4);

__device__ const uint visualityAmplifyCoeff = 10;

template <class Real> __device__ __forceinline__
void fractalRenderMain(pixel_info_t** output, long outputPitch, Pointi outputSize, Rectangle<Real> image, uint maxIterations, uint maxSuperSampling, uint flags)
{
  const Pointi idx = getImageIndexes();
  if(idx.x >= outputSize.x || idx.y >= outputSize.y) return;
  if(idx.x == 0 && idx.y == 0){
    // printf("fractal render main\n");
    // printf("image:\t%f\t%f\t%f\t%f\n", image.left_bottom.x, image.left_bottom.y, image.right_top.x, image.right_top.y);
    // printf("outputsize:\t%d\t%d\n", outputSize.x, outputSize.y);
    // printf("focus:\t%d\t%d\n", focus.x, focus.y);
    // printf("dwell:\t%d\tss:\t%d\n", maxIterations, maxSuperSampling);
    // printf("\n");
  }
  
  //the value of maxSuperSampling will be changed by the calee
  uint result = sampleTheFractal(idx, outputSize, image, maxIterations, maxSuperSampling, flags & USE_ADAPTIVE_SS_FLAG_MASK);

  if(flags & VISUALISE_SAMPLE_COUNT_FLAG_MASK){
    result = maxSuperSampling * visualityAmplifyCoeff;
  }

  pixel_info_t* pOutput = getPtrToPixel(output, outputPitch, idx.x, idx.y);
  pOutput->value = result;
  pOutput->weight = (float) maxSuperSampling;
  ASSERT(pOutput->weight > 0);
}

//section exported global kernels:

extern "C" __global__
void fractalRenderMainFloat(pixel_info_t** output, long outputPitch, Pointi outputSize, Rectangle<float> image, uint maxIterations, uint maxSuperSampling, uint flags){
  fractalRenderMain<float>(output, outputPitch, outputSize, image, maxIterations, maxSuperSampling, flags);
}

extern "C" __global__
void fractalRenderMainDouble(pixel_info_t** output, long outputPitch, Pointi outputSize, Rectangle<double> image, uint maxIterations, uint maxSuperSampling, uint flags){
  fractalRenderMain<double>(output, outputPitch, outputSize, image, maxIterations, maxSuperSampling, flags);

}

extern "C" __global__
void compose(pixel_info_t** inputMain, long inputMainPitch, pixel_info_t** inputBcg, long inputBcgPitch, hipSurfaceObject_t surfaceOutput, uint width, uint height, hipSurfaceObject_t colorPalette, uint paletteLength){
  const uint idx_x = blockDim.x * blockIdx.x + threadIdx.x;
  const uint idx_y = blockDim.y * blockIdx.y + threadIdx.y;
  if(idx_x >= width || idx_y >= height) return;

  /*
  const uint blurSize = 4;
  
  const uint convolution[blurSize][blurSize] = {
      //  {1,2,1},
      //  {2,4,2},
      //  {1,2,1}
      {0,0,0},
      {0,1,0},
      {0,0,0}
  };
  const uint convolutionDivisor = 1;

  uint sum = 0;
  #pragma unroll
  for(uint i = -blurSize/2; i < blurSize/2; i++){ 
    #pragma unroll
    for(uint j = -blurSize/2; j < blurSize/2; j++){
      uint x = max(0,min(width,idx_x + i));
      uint y = max(0,min(height,idx_y + j));
      uint* pInput1 = (uint*)((char*)input1 + y * input1pitch) + x;
      sum += (*pInput1) * convolution[i+blurSize/2][j+blurSize/2];
    }
  }
  uint result;
  result = sum / convolutionDivisor;
  */
  //choose result from one or two

  pixel_info_t* pResult;
  pResult = getPtrToPixel(inputMain, inputMainPitch, idx_x, idx_y);
  uint result = pResult->value;

  uint paletteIdx = paletteLength - (result % paletteLength) - 1;
//  ASSERT(paletteIdx >=0);
  ASSERT(paletteIdx < paletteLength);
  uint resultColor;
  surf2Dread(&resultColor, colorPalette, paletteIdx * sizeof(uint), 0);
  // if(result == maxIterations || result == maxIterations-1)
  //   resultColor = ColorsARGB::YELLOW;

  surf2Dwrite(resultColor, surfaceOutput, idx_x * sizeof(uint), idx_y);
}

extern "C" __global__
void fractalRenderUnderSampled(pixel_info_t** output, long outputPitch, uint width, uint height, float left_bottom_x, float left_bottom_y, float right_top_x, float right_top_y, uint maxIterations, uint underSamplingLevel)
{
  //work only at every Nth pixel:
  const uint idx_x = (blockDim.x * blockIdx.x + threadIdx.x) * underSamplingLevel;
  const uint idx_y = (blockDim.y * blockIdx.y + threadIdx.y) * underSamplingLevel;
  if(idx_x >= width-underSamplingLevel || idx_y >= height-underSamplingLevel) return;
  
  //We are in a complex plane from (left_bottom) to (right_top), so we scale the pixels to it
  float pixelWidth = (right_top_x - left_bottom_x) / (float) width;
  float pixelHeight = (right_top_y - left_bottom_y) / (float) height;
  
  float cx = left_bottom_x + (idx_x)  * pixelWidth;
  float cy = right_top_y - (idx_y) * pixelHeight;

  uint escapeTime = escape(maxIterations, Pointf(cx, cy));

  for(uint x = 0; x < underSamplingLevel; x++){
    for(uint y = 0; y < underSamplingLevel; y++){
      //surf2Dwrite(resultColor, surfaceOutput, (idx_x + x) * sizeof(unsigned uint), (idx_y+y));
      pixel_info_t* pOutput = getPtrToPixel(output, outputPitch, idx_x+x, idx_y+y);
      pOutput->value = escapeTime;
      pOutput->weight = 1 / (float) underSamplingLevel;
    }
  }

}

struct big{
  uint a;
  uint b;
  uint c;
  uint d;
  uint e;
  uint f;
};

extern "C" __global__
void debug(big a, uint c){
  const uint idx_x = blockDim.x * blockIdx.x + threadIdx.x;
  const uint idx_y = blockDim.y * blockIdx.y + threadIdx.y;
  if(idx_x == 0 && idx_y == 0){
    // printf("aa:\t%u\n",a.a);
    // printf("ab:\t%u\n",a.b);
    // printf("ac:\t%u\n",a.c);
    // printf("ad:\t%u\n",a.d);
    // printf("ae:\t%u\n",a.e);
    // printf("af:\t%u\n",a.f);
    // printf("c:\t%u\n",c);
  }
}

extern "C" __global__
void init(){

}

/// for given point <code>p</code> in the current image and given warping information, find cooridnates of the same point (=representing the same point in the fractal's complex plane) in the image being warped
/// @param p: the point whose warping origin is being returned
/// @param imageSize: width and height (in pixels) of current image
/// @param currentImage: rectangle representing the part of the complex plane that is being rendered
/// @param oldImage: rectangle representing the part of the complex plane that is being reused

template <class Real> __device__ __forceinline__
Point<Real> getWarpingOrigin(Point<Real> p, Point<Real> imageSize, Rectangle<Real> currentImage, Rectangle<Real> oldImage){

      Point<Real> size_current = currentImage.size();
      Point<Real> size_reused = oldImage.size();
      Point<Real> coeff = size_current / size_reused;

      Point<Real> deltaReal;    
      deltaReal.x = currentImage.left_bottom.x - oldImage.left_bottom.x;
      deltaReal.y = oldImage.right_top.y - currentImage.right_top.y;
      Point<Real> delta = deltaReal / size_current * imageSize;

      Point<Real> result = (p * coeff) + delta;
      return result;
}

__device__ float screenDistance = 60; //in cm; better be set by the user
__device__ __forceinline__
/// Returns how many samples this pixel should take, based on foveation.
/// Value is between 0 and maxSuperSamplingLevel.
/// Value in the focus will always be maxSuperSamplingLevel, values in the non-peripheral view will always be non-zero.
/// Returned value is the same for all pixels within a warp (the highest is taken).
uint getAdvisedSampleCount(Pointi pixel, Pointi focus, uint maxSuperSamplingLevel){
  //per-warp normalisation, i.e. set all pixels from a warp to same value
  pixel = pixel - (pixel % Pointi(WARP_SIZE_X, WARP_SIZE_Y));

  float pixelRealWidthInCm = 0.02652; //todo this value should probably be entered by the user. From http://www.prismo.ch/comparisons/desktop.php 
  float focusDistance = focus.cast<float>().distanceTo(pixel.cast<float>()) * pixelRealWidthInCm; //distance to focus, translated to cm
  /// visual angle for one eye, i.e possible values are from 0 to ~ 110
  float visualAngle = atan (focusDistance / screenDistance) * 180 / PI_F; //from https://en.wikipedia.org/wiki/Visual_angle
  
  //used model for (visualAngle -> relativeQuality): in (0,fovealViewLimit): a constant function that yields 1, in (fovealViewLimit, peripheralViewLimit): descenidng linear function from 1 to 0
  ASSERT(visualAngle >= 0); //arctan is positive for positive input and distance is always non-negative
  const float fovealViewLimit = 5.5; //in degrees, value from https://en.wikipedia.org/wiki/Peripheral_vision
  ///todo, this number is based on a paper. Based on my experience, it could be even smaller
  const float peripheralViewTreshold = 60;  //in degrees, value from https://en.wikipedia.org/wiki/Peripheral_vision
  float relativeQuality = (1/(fovealViewLimit-peripheralViewTreshold))*visualAngle+(-peripheralViewTreshold/(fovealViewLimit-peripheralViewTreshold)); 
  if(visualAngle <= fovealViewLimit) relativeQuality = 1;

  uint result = maxSuperSamplingLevel * relativeQuality;
  if(visualAngle <= peripheralViewTreshold)
    result = max(1, result); //always return at least 1 for pixels within the field of view
  return result;
}

extern "C" __global__
void fractalRenderReuseSamples(pixel_info_t** output, long outputPitch, Pointi outputSize, Rectangle<float> image, uint maxIterations, uint maxSuperSampling, uint flags, Rectangle<float> imageReused, pixel_info_t** input, long inputPitch, Pointi focus){

  const Pointi idx = getImageIndexes();
  if(idx.x >= outputSize.x || idx.y >= outputSize.y) return;
  if(idx.x == 0 && idx.y == 0){
    // printf("fractalRenderReuseSamples:\n");
    // printf("\n");
  }
  ASSERT(idx.x < outputSize.x);
  ASSERT(idx.y < outputSize.y);

  //sample reusal:
  bool reusingSample = false;
  uint reusalResult;
  float reusalWeight;
  if(flags & USE_SAMPLE_REUSE_FLAG_MASK){
    const Pointf originf = getWarpingOrigin(Pointf(idx.x, idx.y),outputSize.cast<float>(),image, imageReused);
    const Point<int> origin = Point<int>((int)round(originf.x), (int)round(originf.y)); //it is important to convert to signed int, not uint (because the value may be negative)
    if(origin.x < 0 || origin.x >= outputSize.x || origin.y < 0 || origin.y >= outputSize.y){
      reusingSample = false;
    }
    else{
      pixel_info_t* pInput = getPtrToPixel(input, inputPitch, origin.x, origin.y);
      reusalResult = pInput->value;
      reusalWeight = (float) pInput->weight;
      reusingSample = true;
    }
  }
  
  //sample generation:
  uint sampleCount = maxSuperSampling;
  if (flags & USE_FOVEATION_FLAG_MASK)
    sampleCount = getAdvisedSampleCount(idx, focus, maxSuperSampling);
  if((!reusingSample || reusalWeight == 0) && sampleCount == 0)  sampleCount = 1; //at least one sample has to be taken somewhere
      //indeed, it may happen that reusalWeight == 0, because reusalWeight decreases in time (see reusedSampleDegradateCoeff)
  uint renderResult = sampleTheFractal(idx, outputSize, image, maxIterations, sampleCount, flags & USE_ADAPTIVE_SS_FLAG_MASK);
  ASSERT(reusingSample || sampleCount > 0);

  //combine reused and generated samples:
  uint result;
  float resultWeight;
  if(reusingSample){
    if(sampleCount == 0){
      result = reusalResult;
      resultWeight = reusalWeight;    
    }else{
      const float reusedSampleDegradateCoeff = 0.6; // must be <=1
      if(flags & IS_ZOOMING_FLAG_MASK)
        reusalWeight *= reusedSampleDegradateCoeff;
      resultWeight = reusalWeight + sampleCount;    
      result = (reusalResult * reusalWeight + renderResult * sampleCount) / (resultWeight);
    }
  }else{
    result = renderResult;
    resultWeight = sampleCount;
  }

// //debug pravy dolni roh:
//       uint qqq = 16;
//   if(idx.x >= width-qqq || idx.y >= height-qqq)      
//       result = maxIterations;

  if(flags & VISUALISE_SAMPLE_COUNT_FLAG_MASK){
    result = sampleCount * visualityAmplifyCoeff;
  }

  pixel_info_t* pOutput = getPtrToPixel(output, outputPitch, idx.x, idx.y);
  pOutput->value = result;
  pOutput->weight = resultWeight;
  ASSERT(resultWeight > 0);
}


__device__ void printParams_debug(hipSurfaceObject_t surfaceOutput, long outputDataPitch_debug, uint width, uint height, float left_bottom_x, float left_bottom_y, float right_top_x, float right_top_y, uint maxIterations, uint** outputData_debug, hipSurfaceObject_t colorPalette, uint paletteLength, float* randomSamples, uint superSamplingLevel, bool adaptiveSS, bool visualiseSS){
  const uint idx_x = blockDim.x * blockIdx.x + threadIdx.x;
  const uint idx_y = blockDim.y * blockIdx.y + threadIdx.y;
  if(idx_x != 0 || idx_y != 0)
    return;
  printf("\n");
  printf("width:\t%u\n",width);
  printf("height:\t%u\n",height);
  printf("maxIterations:\t%u\n",maxIterations);
  printf("SS lvl:\t%u\n",superSamplingLevel);
}