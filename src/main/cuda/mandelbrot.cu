#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "math.h"
#include "helpers.hpp"
#include "float.h"

typedef unsigned int uint;
using Pointf = Point<float>;
using Pointi = Point<uint>;

__device__ const float PI_F = 3.14159265358979f;
__device__ const uint MAX_SS_LEVEL = 256;

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
  printf("Mandelbrot: Error at %s:%d\n",__FILE__,__LINE__); \
  return EXIT_FAILURE;}} while(0)
  
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
  printf("Mandelbrot: Error at %s:%d\n",__FILE__,__LINE__); \
  return EXIT_FAILURE;}} while(0)

#define DEBUG_MODE
#ifdef DEBUG_MODE 
  #define ASSERT(x) assert(x)
#else 
  #define ASSERT(x) do {} while(0)
#endif

#ifndef CUDART_VERSION
  #error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 9000) //for cuda 9 and later, use __any_sync(__activemask(), predicate) instead, see Programming guide, B.13 for more details
  #define __ALL(predicate) __all_sync(__activemask(), predicate)
  #define __ANY(predicate) __any_sync(__activemask(), predicate)
#else
  #define __ALL(predicate) __all(predicate)
  #define __ANY(predicate) __any(predicate)
#endif



//Mandelbrot content, using standard mathematical terminology for Mandelbrot set definition, i.e.
//  f_n = f_{n-1}^2 + c
//  f_0 = 0
//  thus iteratively applying: f(z) = z*z * c
//  where z, c are complex numbers, with components denoted as
//    x ... for real part (corresponding to geometric x-axis)
//    y ... for imag part (corresponding to geometric y-axis)

template <class Real> __device__ __forceinline__
uint escape(uint dwell, Point<Real> c){
  Point<Real> z(0,0);
  Real zx_new;
  uint i = 0;
  while(i < dwell && z.x*z.x+z.y*z.y < 4){
      zx_new = z.x*z.x-z.y*z.y + c.x;
      z.y = 2*z.x*z.y + c.y; 
      z.x = zx_new;
      ++i;
  }
  return i;
}


/// Dispersion in this context is "Index of dispersion", aka variance-to-mean ratio. See https://en.wikipedia.org/wiki/Index_of_dispersion for more details
template <class Real> __device__ __forceinline__
Real computeDispersion(uint* data, uint dataLength, Real mean){
  uint n = dataLength;
  Real variance = 0;
  for(uint i = 0; i < dataLength; i++){
    //using numerically stable Two-Pass algorithm, https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Two-pass_algorithm
    variance += (data[i]-mean)*(data[i]-mean);
  }
  variance /= (n-1); 
  return variance / mean;
}


__device__ __forceinline__
bool isWithinRadius(uint idx_x, uint idx_y, uint width, uint height, uint radius, uint focus_x, uint focus_y){
  if(__sad(idx_x, focus_x, 0) > radius / 2) return false;
  if(__sad(idx_y, focus_y, 0) > radius / 2) return false;
  else return true;

  // if(idx_x < (width - radius)/2 || idx_y < (height-radius)/2) return false;
  // if(idx_x > (width + radius)/2 || idx_y > (height+radius)/2) return false;
  // else return true;
}

__device__ long seed;
/// Intended for debugging only
__device__ __forceinline__ uint simpleRandom(uint val){
    long long a = 1103515245;
    long long c = 12345;
    long long m = 4294967295l; //2**32 - 1
    seed = (a * (val+seed) + c) % m;
    return seed;
}

__device__ const uint WARP_SIZE_X = 8; //represents desired size of the recatangular warp (with respect to threadIdx). WARP_SIZE.x * WARP_SIZE.y should always be warpSize (32 for CUDA 9 and lower)
__device__ const uint WARP_SIZE_Y = 4;
  /// Computes indexes to a per-pixel acces of a 2D image, based on threadIdx and blockIdx.
  /// Morover, threads in a warp will be arranged in a rectangle (rather than in single line as with the naive implementation).
  /// The caller should always check if the returned value exceeded image width and height.
__device__ const Point<uint> getImageIndexes(){
  const uint threadID = threadIdx.x + threadIdx.y * blockDim.x;
  const uint warpH = WARP_SIZE_Y; // 2,4,8 are only reasonable values of warpH for the following formula
  const uint blockWidth = blockDim.x * warpH;
  ASSERT (blockDim.x == 32); //following formula works only when blockDim.x is 32 
  const uint inblock_idx_x = (-(threadID % (warpH * warpH)) + threadID % blockWidth) / warpH + threadID % warpH;
  const uint inblock_idx_y = (threadID / blockWidth) * warpH + (threadID / warpH) % warpH;
  const uint idx_x = blockDim.x * blockIdx.x + inblock_idx_x;
  const uint idx_y = blockDim.y * blockIdx.y + inblock_idx_y;
  // { //debug
  //   uint warpid = threadID / warpH;
  //   if(idx_x < 20 && idx_y < 20){
  //     //printf("bw:%u\n", blockWidth);
  //     printf("%u\t%u\t%u\t%u\t%u\n", threadIdx.x, threadIdx.y, threadID ,inblock_idx_x, inblock_idx_y);
  //   }
  // }
  return Point<uint>(idx_x, idx_y);
}

__device__ __forceinline__
pixel_info_t* getPtrToPixel(uint** array2D, long pitch, uint x, uint y){
  return (((pixel_info_t*)((char*)array2D + y * pitch)) + x);
}

/// param sampleCount: Maximum number of samples to take. Actual number of samples taken will be stored here before returning. If adaptiveSS==false, the value will not change.
template <class Real> __device__
uint sampleTheFractal(Pointi pixel, uint width, uint height, Rectangle<Real> image, uint dwell,uint & sampleCount, bool adaptiveSS, bool visualiseSS){
  const uint adaptiveTreshold = 10;
  uint r[adaptiveTreshold];

  //We are in a complex plane from (left_bottom) to (right_top), so we scale the pixels to it
  Point<Real> pixelSize = image.size() / Point<Real>(width, height);
  
  uint escapeTimeSum = 0;
  ASSERT (sampleCount <= MAX_SS_LEVEL);
  for(uint i = 0; i < sampleCount; i++){
    Point<Real> delta = Point<Real>(i / (Real) sampleCount);
    
    // c = {LBx, RTy} {+,-} ((pixel+delta) * pixelSize)
    const Point<Real> c = Point<Real>(image.left_bottom.x, image.right_top.y) +
      Point<Real>(1,-1) * (pixel.cast<Real>() + delta) * pixelSize;

    uint escapeTime = escape(dwell, c);
    escapeTimeSum += escapeTime;
    if(i < adaptiveTreshold){
      r[i] = escapeTime;
    }

    //todo this process should ideally adaptive, at least dynamic. Not predefined like.
    if(( i== 2 || i == adaptiveTreshold) && adaptiveSS){ //decide whether to continue with supersampling or not
      Real mean = escapeTimeSum / (i+1);
      Real dispersion = computeDispersion(r, i, mean);
      float dispersionMax = (i==2) ? 0.01 : 0.1;
      if(__ALL(dispersion <= dispersionMax )){
        sampleCount = i+1; //effectively disabling high SS and storing info about actual number of samples taken
      }
      //else we are on an chaotic edge, thus as many samples as possible are needed
    }
  }
  uint result = escapeTimeSum / sampleCount; 
  return result;
   

  //debug:
  // if(idx_x < 10 && idx_y < 10){
  //   printf("%f\t", randomSample);
  //   __syncthreads();
  //   if(idx_x == 0 && idx_y == 0)
  //     printf("\n");
  // }
} 

template <class Real> __device__ __forceinline__
void fractalRenderMain(uint** output, long outputPitch, uint width, uint height, Rectangle<Real> image, uint dwell, uint superSamplingLevel, bool adaptiveSS, bool visualiseSS, float* randomSamples, uint renderRadius, uint focus_x, uint focus_y, bool isDoublePrecision)
// todo: usporadat poradi paramateru, cudaXXObjects predavat pointrem, ne kopirovanim (tohle rozmyslet, mozna je to takhle dobre)
//  todo ma to fakt hodne pointeru, mnoho z nich je pritom pro vsechny launche stejny - nezdrzuje tohle? omezene registry a tak
{
  const Pointi idx = getImageIndexes();
  if(idx.x >= width || idx.y >= height) return;
  // if(idx.x == 0 && idx.y == 0){
  //   printf();
  // }
  //if(!isWithinRadius(idx.x, idx.y, width, height, renderRadius, focus_x, focus_y)) return;
  
  uint result = sampleTheFractal(idx, width, height, image, dwell, superSamplingLevel, adaptiveSS, visualiseSS);

  pixel_info_t* pOutput = getPtrToPixel(output, outputPitch, idx.x, idx.y);
  pOutput->value = result;
  pOutput->weight = (float) superSamplingLevel;
  ASSERT(pOutput->weight > 0);
}

//section exported global kernels:

extern "C" __global__
void fractalRenderMainFloat(uint** output, long outputPitch, uint width, uint height, float left_bottom_x, float left_bottom_y, float right_top_x, float right_top_y, uint dwell, uint superSamplingLevel, bool adaptiveSS, bool visualiseSS, float* randomSamples, uint renderRadius, uint focus_x, uint focus_y){
  fractalRenderMain<float>(output, outputPitch, width, height, Rectangle<float>(left_bottom_x, left_bottom_y, right_top_x, right_top_y), dwell, superSamplingLevel, adaptiveSS, visualiseSS, randomSamples,  renderRadius, focus_x, focus_y, false);
}

extern "C" __global__
void fractalRenderMainDouble(uint** output, long outputPitch, uint width, uint height, double left_bottom_x, double left_bottom_y, double right_top_x, double right_top_y, uint dwell, uint superSamplingLevel, bool adaptiveSS, bool visualiseSS, float* randomSamples, uint renderRadius, uint focus_x, uint focus_y){
  fractalRenderMain<double>(output, outputPitch, width, height, Rectangle<double>(left_bottom_x, left_bottom_y, right_top_x, right_top_y), dwell, superSamplingLevel, adaptiveSS, visualiseSS, randomSamples,  renderRadius, focus_x, focus_y, true);

}

extern "C" __global__
void compose(uint** inputMain, long inputMainPitch, uint** inputBcg, long inputBcgPitch, hipSurfaceObject_t surfaceOutput, uint width, uint height, hipSurfaceObject_t colorPalette, uint paletteLength, uint dwell, uint mainRenderRadius, uint focus_x, uint focus_y){
  const uint idx_x = blockDim.x * blockIdx.x + threadIdx.x;
  const uint idx_y = blockDim.y * blockIdx.y + threadIdx.y;
  if(idx_x >= width || idx_y >= height) return;

  /*
  const uint blurSize = 4;
  
  const uint convolution[blurSize][blurSize] = {
      //  {1,2,1},
      //  {2,4,2},
      //  {1,2,1}
      {0,0,0},
      {0,1,0},
      {0,0,0}
  };
  const uint convolutionDivisor = 1;

  uint sum = 0;
  #pragma unroll
  for(uint i = -blurSize/2; i < blurSize/2; i++){ 
    #pragma unroll
    for(uint j = -blurSize/2; j < blurSize/2; j++){
      uint x = max(0,min(width,idx_x + i));
      uint y = max(0,min(height,idx_y + j));
      uint* pInput1 = (uint*)((char*)input1 + y * input1pitch) + x;
      sum += (*pInput1) * convolution[i+blurSize/2][j+blurSize/2];
    }
  }
  uint result;
  result = sum / convolutionDivisor;
  */
  //choose result from one or two

  pixel_info_t* pResult;
  if(isWithinRadius(idx_x, idx_y, width, height, mainRenderRadius, focus_x, focus_y)){
    pResult = getPtrToPixel(inputMain, inputMainPitch, idx_x, idx_y);
  }else{
    pResult = getPtrToPixel(inputBcg, inputBcgPitch, idx_x, idx_y);
  }
  uint result = pResult->value;

  uint paletteIdx = paletteLength - (result % paletteLength) - 1;
//  ASSERT(paletteIdx >=0);
  ASSERT(paletteIdx < paletteLength);
  uint resultColor;
  surf2Dread(&resultColor, colorPalette, paletteIdx * sizeof(uint), 0);
  // if(result == dwell || result == dwell-1)
  //   resultColor = ColorsARGB::YELLOW;

  surf2Dwrite(resultColor, surfaceOutput, idx_x * sizeof(uint), idx_y);
}

extern "C" __global__
void blur(){}

extern "C" __global__
void fractalRenderUnderSampled(uint** output, long outputPitch, uint width, uint height, float left_bottom_x, float left_bottom_y, float right_top_x, float right_top_y, uint dwell, uint underSamplingLevel)
{
  //work only at every Nth pixel:
  const uint idx_x = (blockDim.x * blockIdx.x + threadIdx.x) * underSamplingLevel;
  const uint idx_y = (blockDim.y * blockIdx.y + threadIdx.y) * underSamplingLevel;
  if(idx_x >= width-underSamplingLevel || idx_y >= height-underSamplingLevel) return;
  
  //We are in a complex plane from (left_bottom) to (right_top), so we scale the pixels to it
  float pixelWidth = (right_top_x - left_bottom_x) / (float) width;
  float pixelHeight = (right_top_y - left_bottom_y) / (float) height;
  
  float cx = left_bottom_x + (idx_x)  * pixelWidth;
  float cy = right_top_y - (idx_y) * pixelHeight;

  uint escapeTime = escape(dwell, Pointf(cx, cy));

  for(uint x = 0; x < underSamplingLevel; x++){
    for(uint y = 0; y < underSamplingLevel; y++){
      //surf2Dwrite(resultColor, surfaceOutput, (idx_x + x) * sizeof(unsigned uint), (idx_y+y));
      pixel_info_t* pOutput = getPtrToPixel(output, outputPitch, idx_x+x, idx_y+y);
      pOutput->value = escapeTime;
      pOutput->weight = 1 / (float) underSamplingLevel;
    }
  }

}

struct big{
  uint a;
  uint b;
  uint c;
  uint d;
  uint e;
  uint f;
};

extern "C" __global__
void debug(big a, uint c){
  const uint idx_x = blockDim.x * blockIdx.x + threadIdx.x;
  const uint idx_y = blockDim.y * blockIdx.y + threadIdx.y;
  if(idx_x == 0 && idx_y == 0){
    // printf("aa:\t%u\n",a.a);
    // printf("ab:\t%u\n",a.b);
    // printf("ac:\t%u\n",a.c);
    // printf("ad:\t%u\n",a.d);
    // printf("ae:\t%u\n",a.e);
    // printf("af:\t%u\n",a.f);
    // printf("c:\t%u\n",c);
  }
}

extern "C" __global__
void init(){

}

/// for given point <code>p</code> in the current image and given warping information, find cooridnates of the same point (=representing the same point in the fractal's complex plane) in the image being warped
/// @param p: the point whose warping origin is being returned
/// @param imageSize: width and height (in pixels) of current image
/// @param currentImage: rectangle representing the part of the complex plane that is being rendered
/// @param oldImage: rectangle representing the part of the complex plane that is being reused

template <class Real> __device__ __forceinline__
Point<Real> getWarpingOrigin(Point<Real> p, Point<Real> imageSize, Rectangle<Real> currentImage, Rectangle<Real> oldImage){

      Point<Real> size_current = currentImage.size();
      Point<Real> size_reused = oldImage.size();
      Point<Real> coeff = size_current / size_reused;

      Point<Real> deltaReal;    
      deltaReal.x = currentImage.left_bottom.x - oldImage.left_bottom.x;
      deltaReal.y = oldImage.right_top.y - currentImage.right_top.y;
      Point<Real> delta = deltaReal / size_current * imageSize;

      Point<Real> result = (p * coeff) + delta;
      return result;
}

__device__ float screenDistance = 60; //in cm; better be set by the user
__device__ __forceinline__
/// Returns how many samples this pixel should take, based on foveation.
/// Value is between 0 and maxSuperSamplingLevel.
/// Value in the focus will always be maxSuperSamplingLevel, values in the non-peripheral view will always be non-zero.
/// Returned value is the same for all pixels within a warp (the highest is taken).
uint getAdvisedSampleCount(Pointi pixel, Pointi focus, uint maxSuperSamplingLevel){
  //per-warp normalisation, i.e. set all pixels from a warp to same value
  pixel = pixel - (pixel % Pointi(WARP_SIZE_X, WARP_SIZE_Y));

  float pixelRealWidthInCm = 0.02652; //todo this value should probably be entered by the user. From http://www.prismo.ch/comparisons/desktop.php 
  float focusDistance = focus.cast<float>().distanceTo(pixel.cast<float>()) * pixelRealWidthInCm; //distance to focus, translated to cm
  /// visual angle for one eye, i.e possible values are from 0 to ~ 110
  float visualAngle = atan (focusDistance / screenDistance) * 180 / PI_F; //from https://en.wikipedia.org/wiki/Visual_angle
  
  //used model for (visualAngle -> relativeQuality): in (0,fovealViewLimit): a constant function that yields 1, in (fovealViewLimit, peripheralViewLimit): descenidng linear function from 1 to 0
  ASSERT(visualAngle >= 0); //arctan is positive for positive input and distance is always non-negative
  const float fovealViewLimit = 5.5; //in degrees, value from https://en.wikipedia.org/wiki/Peripheral_vision
  ///todo, this number is based on a paper. Based on my experience, it could be even smaller
  const float peripheralViewTreshold = 60;  //in degrees, value from https://en.wikipedia.org/wiki/Peripheral_vision
  float relativeQuality = (1/(fovealViewLimit-peripheralViewTreshold))*visualAngle+(-peripheralViewTreshold/(fovealViewLimit-peripheralViewTreshold)); 
  if(visualAngle <= fovealViewLimit) relativeQuality = 1;

  uint result = maxSuperSamplingLevel * relativeQuality;
  if(visualAngle <= peripheralViewTreshold)
    result = max(1, result); //always return at least 1 for pixels within the field of view
  return result;
}

extern "C" __global__
void fractalRenderReuseSamples(uint** output, long outputPitch, uint width, uint height, float a, float b, float c, float d, uint dwell, uint superSamplingLevel, bool adaptiveSS, bool visualiseSS, float* randomSamples, uint renderRadius, uint focus_x, uint focus_y, float p, float q, float r, float s, uint** input, long inputPitch, bool useFoveation, bool useSampleReuse){

  const Pointi idx = getImageIndexes();
  if(idx.x >= width || idx.y >= height) return;
  if(idx.x == 0 && idx.y == 0){
    printf("fractalRenderReuseSamples:\n");
  }
  ASSERT(idx.x < width);
  ASSERT(idx.y < height);
  Rectangle<float> image = Rectangle<float>(a,b,c,d);

  useSampleReuse = true;
  useFoveation = true;

  //sample reusal:
  bool reusingSample = false;
  uint reusalResult;
  float reusalWeight;
  if(useSampleReuse){
    const Pointf originf = getWarpingOrigin(Pointf(idx.x, idx.y),Pointf(width,height),image, Rectangle<float>(p,q,r,s));
    const Point<int> origin = Point<int>((int)round(originf.x), (int)round(originf.y)); //it is important to convert to signed int, not uint (because the value may be negative)
    if(origin.x < 0 || origin.x >= width || origin.y < 0 || origin.y >= height){
      reusalResult = 404;   //not-found error :)
      reusingSample = false;
    }
    else{
      pixel_info_t* pInput = getPtrToPixel(input, inputPitch, origin.x, origin.y);
      reusalResult = pInput->value;
      reusalWeight = (float) pInput->weight;
      reusingSample = true;
    }
  }
  
  //sample generation:
  uint sampleCount = superSamplingLevel;
  if (useFoveation)
    sampleCount = getAdvisedSampleCount(idx, Pointi(focus_x, focus_y), superSamplingLevel);
  if((!reusingSample || reusalWeight == 0) && sampleCount == 0)  sampleCount = 1; //at least one sample has to be taken somewhere
      //indeed, it may happen that reusalWeight == 0, because reusalWeight decreases in time (see reusedSampleDegradateCoeff)
  uint renderResult = sampleTheFractal(idx, width, height, image, dwell, sampleCount, adaptiveSS, visualiseSS);
  ASSERT(reusingSample || sampleCount > 0);

  //combine reused and generated samples:
  uint result;
  float resultWeight;
  if(reusingSample){
    if(sampleCount == 0){
      result = reusalResult;
      resultWeight = reusalWeight;    
    }else{
      const float reusedSampleDegradateCoeff = 0.65; // must be <=1
      reusalWeight *= reusedSampleDegradateCoeff;
      resultWeight = reusalWeight + sampleCount;    
      result = (reusalResult * reusalWeight + renderResult * sampleCount) / (resultWeight);
    }
  }else{
    result = renderResult;
    resultWeight = sampleCount;
  }

// //debug pravy dolni roh:
//       uint qqq = 16;
//   if(idx.x >= width-qqq || idx.y >= height-qqq)      
//       result = dwell;

  if(visualiseSS){
    if(reusingSample) ++sampleCount;
    result = sampleCount / float (superSamplingLevel) * 255;
  }

  pixel_info_t* pOutput = getPtrToPixel(output, outputPitch, idx.x, idx.y);
  pOutput->value = result;
  pOutput->weight = resultWeight;
  ASSERT(resultWeight > 0);
}


__device__ void printParams_debug(hipSurfaceObject_t surfaceOutput, long outputDataPitch_debug, uint width, uint height, float left_bottom_x, float left_bottom_y, float right_top_x, float right_top_y, uint dwell, uint** outputData_debug, hipSurfaceObject_t colorPalette, uint paletteLength, float* randomSamples, uint superSamplingLevel, bool adaptiveSS, bool visualiseSS){
  const uint idx_x = blockDim.x * blockIdx.x + threadIdx.x;
  const uint idx_y = blockDim.y * blockIdx.y + threadIdx.y;
  if(idx_x != 0 || idx_y != 0)
    return;
  printf("\n");
  printf("width:\t%u\n",width);
  printf("height:\t%u\n",height);
  printf("dwell:\t%u\n",dwell);
  printf("SS lvl:\t%u\n",superSamplingLevel);
}