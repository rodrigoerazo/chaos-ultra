#include "hip/hip_runtime.h"
#include "fractal.cuh"

__constant__ double julia_c[2]; 

template <class Real> __device__ __forceinline__
float computeFractal(unsigned int maxIterations, Point<Real> z){
  Point<Real> c((Real) julia_c[0],(Real) julia_c[1]);
  Real zx_new;
  unsigned int i = 0;
  while(i < maxIterations && z.x*z.x+z.y*z.y < 4){
      zx_new = z.x*z.x-z.y*z.y + c.x;
      z.y = 2*z.x*z.y + c.y; 
      z.x = zx_new;
      ++i;
  }
  return i;
}

__device__ __forceinline__
unsigned int colorize(hipSurfaceObject_t colorPalette, unsigned int paletteLength, float iterationResult){
  unsigned int iterationResult_i = round(iterationResult);
  unsigned int paletteIdx = paletteLength - (iterationResult_i % paletteLength) - 1;
  ASSERT(paletteIdx < paletteLength);
  unsigned int resultColor;
  surf2Dread(&resultColor, colorPalette, paletteIdx * sizeof(unsigned int), 0);
  return resultColor;
}

__device__ void debugFractal(){
  printf("hello from julia\n");
}